
#include <hip/hip_runtime.h>
#define N 10
#define THREADS_PER_BLOCK 20

#ifdef DEBUG
#include "stdio.h"
#endif

__global__ void matmul(int *a, int *b, int *c) {
  int c_idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (c_idx >= N * N) {
    return;
  }

  int *a_row = a + (c_idx / N) * N;
  int *b_col = b + c_idx % N;
  int *c_write = c + c_idx;

  for (int k = 0; k < N; ++k)
    *c_write += a_row[k] * b_col[k * N];
}

int main(void) {
  int array_size = N * N;
  int array_memory_size = array_size * sizeof(int);

  int *a = (int *)malloc(array_memory_size);
  for (int i = 0; i < array_size; ++i) {
    if (i % N == 0)
      a[i] = i / N;
    else
      a[i] = 0;
  }
  int *b = (int *)malloc(array_memory_size);
  for (int i = 0; i < array_size; ++i) {
    if (i / N == i % N)
      b[i] = 1;
    else
      b[i] = 0;
  }

#ifdef DEBUG
  printf("a:\n");
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      printf("%d ", a[i * N + j]);
    }
    printf("\n");
  }
  printf("b:\n");
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      printf("%d ", b[i * N + j]);
    }
    printf("\n");
  }
  printf("\n");
#endif

  int *dev_a;
  hipMalloc((void **)&dev_a, array_memory_size);
  hipMemcpy(dev_a, a, array_memory_size, hipMemcpyHostToDevice);

  int *dev_b;
  hipMalloc((void **)&dev_b, array_memory_size);
  hipMemcpy(dev_b, b, array_memory_size, hipMemcpyHostToDevice);

  int *dev_c;
  hipMalloc((void **)&dev_c, array_memory_size);

  matmul<<<(array_size / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK>>>(
      dev_a, dev_b, dev_c);

  int *c = (int *)malloc(array_memory_size);
  hipMemcpy(c, dev_c, array_memory_size, hipMemcpyDeviceToHost);

#ifdef DEBUG
  printf("c:\n");
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      printf("%d ", c[i * N + j]);
    }
    printf("\n");
  }
#endif

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  free(a);
  free(b);
  free(c);

  return 0;
}