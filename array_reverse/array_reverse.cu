#include "hip/hip_runtime.h"
#include "stdio.h"

__global__ void array_reverse(int *a, int *aR) {
  int a_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int temp = a[a_idx];
  aR[N - 1 - a_idx] = temp;
}

int main(void) {
  int array_size = N;
  int array_memory_size = array_size * sizeof(int);

  int *a = (int *)malloc(array_memory_size);
  for (int i = 0; i < array_size; ++i)
    a[i] = i;

#ifdef DEBUG
  printf("Input:\n");
  for (int i = 0; i < N; ++i)
    printf("%d ", a[i]);
  printf("\n");
#endif

  int *dev_a;
  hipMalloc((void **)&dev_a, array_memory_size);
  hipMemcpy(dev_a, a, array_memory_size, hipMemcpyHostToDevice);

  int *dev_aT;
  hipMalloc((void **)&dev_aT, array_memory_size);

  array_reverse<<<N / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(dev_a, dev_aT);

  hipMemcpy(a, dev_aT, array_memory_size, hipMemcpyDeviceToHost);

#ifdef DEBUG
  printf("Output:\n");
  for (int i = 0; i < N; ++i)
    printf("%d ", a[i]);
  printf("\n");
#endif

  hipFree(dev_a);
  hipFree(dev_aT);

  free(a);

  return 0;
}