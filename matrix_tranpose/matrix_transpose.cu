#include "hip/hip_runtime.h"
#include "stdio.h"

// each block takes care of a row of the transposed matrix
__global__ void tranpose_kernel(int *a, int *aT) {
  int aT_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int a_idx = threadIdx.x * gridDim.x + blockIdx.x;
  aT[aT_idx] = a[a_idx];
}

int main(void) {
  int array_size = N * N;
  int array_memory_size = array_size * sizeof(int);

  int *a = (int *)malloc(array_memory_size);
  for (int i = 0; i < array_size; ++i) {
    a[i] = 0;
  }
  a[array_size - N] = 1;

#ifdef DEBUG
  printf("Input:\n");
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      printf("%d ", a[i * N + j]);
    }
    printf("\n");
  }
  printf("\n");
#endif

  int *dev_a;
  hipMalloc((void **)&dev_a, array_memory_size);
  hipMemcpy(dev_a, a, array_memory_size, hipMemcpyHostToDevice);

  int *dev_aT;
  hipMalloc((void **)&dev_aT, array_memory_size);

  tranpose_kernel<<<N, THREADS_PER_BLOCK>>>(dev_a, dev_aT);

  hipMemcpy(a, dev_aT, array_memory_size, hipMemcpyDeviceToHost);

#ifdef DEBUG
  printf("Output:\n");
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      printf("%d ", a[i * N + j]);
    }
    printf("\n");
  }
#endif

  hipFree(dev_a);
  hipFree(dev_aT);

  free(a);

  return 0;
}