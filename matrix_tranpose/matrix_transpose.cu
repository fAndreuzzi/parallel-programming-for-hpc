#include "hip/hip_runtime.h"
#include "stdio.h"

// each block takes care of a row of the transposed matrix
__global__ void tranpose_kernel(int *a) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  int i = idx / N;
  int j = idx % N;
  if (i < j) {
    int temp = a[i * N + j];
    a[i * N + j] = a[j * N + i];
    a[j * N + i] = temp;
  }
}

int main(void) {
  int array_size = N * N;
  int array_memory_size = array_size * sizeof(int);

  int *a = (int *)malloc(array_memory_size);
  for (int i = 0; i < array_size; ++i) {
    a[i] = 0;
  }
  a[array_size - N] = 1;

#ifdef DEBUG
  printf("Input:\n");
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      printf("%d ", a[i * N + j]);
    }
    printf("\n");
  }
  printf("\n");
#endif

  int *dev_a;
  hipMalloc((void **)&dev_a, array_memory_size);
  hipMemcpy(dev_a, a, array_memory_size, hipMemcpyHostToDevice);
  tranpose_kernel<<<N * N / THREADS_PER_BLOCK + 1, THREADS_PER_BLOCK>>>(dev_a);

  hipMemcpy(a, dev_a, array_memory_size, hipMemcpyDeviceToHost);

#ifdef DEBUG
  printf("Output:\n");
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      printf("%d ", a[i * N + j]);
    }
    printf("\n");
  }
#endif

  hipFree(dev_a);
  free(a);

  return 0;
}